#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <tuple>
#include <chrono>

#include "cuda_kernel.h"
#include "modelW.hpp"
#include "utils.cuh"

// TODO add code here

// layer 2 regular_conv2d

__global__ void layer2_gpu_kernel(int *d_cuda_layer_1_output, signed char *d_layer_2_bias, signed char *d_cuda_layer_2_weight, int *d_cuda_layer_2_output){

    int N = (28+1); // +1 to cover all edges (fixes bug #ky2)
    int kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < 1){
            if(m < 32) {
                d_cuda_layer_2_output[index4D_cuda(b,h,w,m,26,26,32)] = d_layer_2_bias[m];
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < 3; kH++) {
            for (int kW = 0; kW < 3; kW++) {
                if(b < 1){
                    for (int c = 0; c < 1; c++) {
                        if(m < 32) {
                            d_cuda_layer_2_output[index4D_cuda(b,h,w,m,26,26,32)] += d_cuda_layer_2_weight[index4D_cuda(kH,kW,c,m,3,1,32)] * d_cuda_layer_1_output[index4D_cuda(b,(h * 1 + kH - 0),(w * 1 + kW - 0),c,28,28,1)];
                        }
                    }
                }
            }
        }
    }
    
}

float layer2_gpu_cuda(int * cuda_layer_1_output, int * cuda_layer_2_output){
    //setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_2_weight
    signed char *cuda_layer_2_weight = (signed char *) layer_2_weight;

    // flatten layer_1_output -> already flattened?
    // int *cuda_layer_1_output = (int *) layer_1_output;

    // prepare for kernel call
    // declare storage on device
    int *d_cuda_layer_1_output; // storage on device for cuda_layer_1_output
    signed char *d_layer_2_bias; // storage on device for layer_2_bias
    signed char *d_cuda_layer_2_weight; // storage on device for cuda_layer_2_weight
    int *d_cuda_layer_2_output; // RESULT storage on device for cuda_layer_2_output

    // allocate GPU device buffers
    // Note: batch_size included in input and output shapes
    hipMalloc((void **) &d_cuda_layer_1_output, 1*1*28*28*sizeof(int)); // dim of cuda_layer_1_output
    hipMalloc((void **) &d_layer_2_bias, 32*sizeof(signed char)); // dim of layer_2_bias
    hipMalloc((void **) &d_cuda_layer_2_weight, 3*3*1*32*sizeof(signed char)); // dim of layer_2_weight
    hipMalloc((void **) &d_cuda_layer_2_output, 1*32*26*26*sizeof(int)); // dim of layer_2_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_1_output, cuda_layer_1_output, (1*1*28*28*sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_2_bias, layer_2_bias, (32*sizeof(signed char)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_2_weight, cuda_layer_2_weight, (3*3*1*32*sizeof(signed char)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 26;
    const int BLKYSIZE = 1;
    const int BLKZSIZE = 1;
    const int GRIDXSIZE = 1;
    const int GRIDYSIZE = 26;
    const int GRIDZSIZE = 32;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer2_gpu_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_1_output, d_layer_2_bias, d_cuda_layer_2_weight, d_cuda_layer_2_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_2_output, d_cuda_layer_2_output, (1*32*26*26*sizeof(int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_1_output);
    hipFree(d_layer_2_bias);
    hipFree(d_cuda_layer_2_weight);
    hipFree(d_cuda_layer_2_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}

// layer 4 maxpool

__global__ void layer4_gpu_kernel(unsigned int *d_cuda_layer_3_output, unsigned int *d_cuda_layer_4_output){
    
    int N = (26+1); // +1 to cover all edges (fixes bug #ky2)
    int kernel_size = 2;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int c = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < 1){
            if(c < 1)
            {
                d_cuda_layer_4_output[index4D_cuda(b,h,w,c,13,13,1)] = 0;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < 2; kH++) {
            for (int kW = 0; kW < 2; kW++) {
                if(b < 1){
                    if(c < 1)
                    {
                        d_cuda_layer_4_output[index4D_cuda(b,h,w,c,13,13,1)] |= d_cuda_layer_3_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,26,26,32)];
                    }
                }
            }
        }
    }

}

float layer4_gpu_cuda(unsigned int * cuda_layer_3_output, unsigned int * cuda_layer_4_output){
    //setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // no 3D arrays to be flattened

    // prepare for kernel call
    // declare storage on device
    unsigned int *d_cuda_layer_3_output; // storage on device for cuda_layer_3_output
    unsigned int *d_cuda_layer_4_output; // RESULT storage on device for cuda_layer_4_output

    // allocate GPU device buffers
    // Note: batch_size included in input and output shapes
    hipMalloc((void **) &d_cuda_layer_3_output, 1*32*26*26*sizeof(unsigned int)); // dim of cuda_layer_3_output
    hipMalloc((void **) &d_cuda_layer_4_output, 1*32*13*13*sizeof(unsigned int)); // dim of layer_4_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_3_output, cuda_layer_3_output, (1*32*26*26*sizeof(unsigned int)), hipMemcpyHostToDevice);

    // define thread and block sizes
    const int BLKXSIZE = 13;
    const int BLKYSIZE = 1;
    const int BLKZSIZE = 1;
    const int GRIDXSIZE = 1;
    const int GRIDYSIZE = 13;
    const int GRIDZSIZE = 1;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer4_gpu_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_3_output, d_cuda_layer_4_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_4_output, d_cuda_layer_4_output, (1*32*13*13*sizeof(unsigned int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_3_output);
    hipFree(d_cuda_layer_4_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}

// layer 5 conv2d

__global__ void layer5_gpu_kernel(unsigned int *d_cuda_layer_4_output, signed char *d_layer_5_bias, unsigned int *d_cuda_layer_5_weight, signed int *d_cuda_layer_5_output){
    
    int N = (13+1); // +1 to cover all edges (fixes bug #ky2)
    int kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < 1){
            if(m < 32) {
                d_cuda_layer_5_output[index4D_cuda(b,h,w,m,11,11,32)] = d_layer_5_bias[m];
            }
        }
    }

    __syncthreads();

    if(idx < N*N){
        for (int kH = 0; kH < 3; kH++) {
            for (int kW = 0; kW < 3; kW++) {
                if(m < 32) {
                    for (int c = 0; c < 1; c++) {
                        d_cuda_layer_5_output[index4D_cuda(b,h,w,m,11,11,32)] += 2 * __popc((unsigned int)~(unsigned int)(d_cuda_layer_5_weight[index4D_cuda(kH,kW,m,c,3,32,1)] ^ d_cuda_layer_4_output[index4D_cuda(b,(h * 1 + kH - 0),(w * 1 + kW - 0),c,13,13,1)])) - 32;
                    }
                }
            }
        }
    }
}

float layer5_gpu_cuda(unsigned int * cuda_layer_4_output, signed int * cuda_layer_5_output){

    //setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_5_weight
    unsigned int *cuda_layer_5_weight = (unsigned int *) layer_5_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned int *d_cuda_layer_4_output; // storage on device for cuda_layer_4_output
    signed char *d_layer_5_bias; // storage on device for layer_5_bias
    unsigned int *d_cuda_layer_5_weight; // storage on device for cuda_layer_5_weight
    signed int *d_cuda_layer_5_output; // RESULT storage on device for cuda_layer_5_output

    // allocate GPU device buffers
    // Note: batch_size included in input and output shapes
    hipMalloc((void **) &d_cuda_layer_4_output, 1*32*13*13*sizeof(unsigned int)); // dim of cuda_layer_4_output
    hipMalloc((void **) &d_layer_5_bias, 32*sizeof(signed char)); // dim of layer_5_bias
    hipMalloc((void **) &d_cuda_layer_5_weight, 3*3*32*32*sizeof(unsigned int)); // dim of layer_5_weight
    hipMalloc((void **) &d_cuda_layer_5_output, 1*32*11*11*sizeof(signed int)); // dim of layer_5_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_4_output, cuda_layer_4_output, (1*32*13*13*sizeof(unsigned int)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_5_bias, layer_5_bias, (32*sizeof(signed char)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_5_weight, cuda_layer_5_weight, (3*3*32*32*sizeof(unsigned int)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 11;
    const int BLKYSIZE = 1;
    const int BLKZSIZE = 1;
    const int GRIDXSIZE = 1;
    const int GRIDYSIZE = 11;
    const int GRIDZSIZE = 32;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer5_gpu_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_4_output, d_layer_5_bias, d_cuda_layer_5_weight, d_cuda_layer_5_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_5_output, d_cuda_layer_5_output, (1*32*11*11*sizeof(signed int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_4_output);
    hipFree(d_layer_5_bias);
    hipFree(d_cuda_layer_5_weight);
    hipFree(d_cuda_layer_5_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}
// layer 7 maxpool

__global__ void layer7_gpu_kernel(unsigned int *d_cuda_layer_6_output, unsigned int *d_cuda_layer_7_output){
    
    int N = (11+1); // +1 to cover all edges (fixes bug #ky2)
    int kernel_size = 2;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int c = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.y + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < 1){
            if(c < 1)
            {
                d_cuda_layer_7_output[index4D_cuda(b,h,w,c,5,5,1)] = 0;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < 2; kH++) {
            for (int kW = 0; kW < 2; kW++) {
                if(b < 1){
                    if(c < 1)
                    {
                        d_cuda_layer_7_output[index4D_cuda(b,h,w,c,5,5,1)] |= d_cuda_layer_6_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,11,11,32)];
                    }
                }
            }
        }
    }

}

float layer7_gpu_cuda(unsigned int * cuda_layer_6_output, unsigned int * cuda_layer_7_output){
    //setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // no 3D arrays to be flattened

    // prepare for kernel call
    // declare storage on device
    unsigned int *d_cuda_layer_6_output; // storage on device for cuda_layer_6_output
    unsigned int *d_cuda_layer_7_output; // RESULT storage on device for cuda_layer_7_output

    // allocate GPU device buffers
    // Note: batch_size included in input and output shapes
    hipMalloc((void **) &d_cuda_layer_6_output, 1*32*11*11*sizeof(unsigned int)); // dim of cuda_layer_6_output
    hipMalloc((void **) &d_cuda_layer_7_output, 1*32*5*5*sizeof(unsigned int)); // dim of layer_7_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_6_output, cuda_layer_6_output, (1*32*11*11*sizeof(unsigned int)), hipMemcpyHostToDevice);

    // define thread and block sizes
    const int BLKXSIZE = 5;
    const int BLKYSIZE = 1;
    const int BLKZSIZE = 1;
    const int GRIDXSIZE = 1;
    const int GRIDYSIZE = 5;
    const int GRIDZSIZE = 1;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer7_gpu_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_6_output, d_cuda_layer_7_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_7_output, d_cuda_layer_7_output, (1*32*5*5*sizeof(unsigned int)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_6_output);
    hipFree(d_cuda_layer_7_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}

// layer 9 gemm
// layer 11 gemm
